/*** fir.gpu.cu**/


#include <hip/hip_runtime.h>
#include <stdio.h>
/** Tool function*/

void random_ints(int* a, int N){
int i;
 for(i = 0; i < N; ++i)
 a[i] = (int)(rand() / (RAND_MAX + 1.0) * 10.0);
}

/** CUDA parameters */

#define BLOCK_SIZE 512

// Define fir coefficients in the texture memory
#define COEF_NBR 5

__constant__ int C[]={1, -2, 4, -2, 1};
int h_C[]={1, -2, 4, -2, 1}; //host fir coefficients
/* * FIR kernel */

__global__ void fir(int *D, int *O) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
int i;
 O[index] = 0;
 for(i=0; i<COEF_NBR; i++)
 O[index] += C[i] * D[index + i];
}

/** FIR kernel that uses shared memory */

__global__ void fir_fast(int *D, int *O) {
int index = threadIdx.x + blockIdx.x * blockDim.x;
__shared__ int tmpD[BLOCK_SIZE + COEF_NBR-1]; // reserve with overlapped data
__shared__ int tmpC[COEF_NBR];
__shared__ int tmpO[BLOCK_SIZE];
int i; //my loop iterator

 // Copy block of input data to shared memory
 tmpD[threadIdx.x] = D[index];
 // Copy overlapped input data
 if( threadIdx.x < COEF_NBR-1 )
 tmpD[blockDim.x + threadIdx.x] = D[(blockIdx.x+1) * blockDim.x + threadIdx.x];
 // Copy fir coefficients
 if( threadIdx.x < COEF_NBR)
 tmpC[threadIdx.x] = C[threadIdx.x];

 __syncthreads();

 tmpO[threadIdx.x] = 0;
 for(i=0; i<COEF_NBR; i++)
 tmpO[threadIdx.x] += tmpC[i] * tmpD[threadIdx.x + i];
 __syncthreads();
  O[index] = tmpO[threadIdx.x];
}

/** host program */

int main(void) {
int *h_D, *h_O; // host copies of D, O
int *d_D, *d_O; // device copies of D, O
int nBlk = 512;
int nThx = BLOCK_SIZE;
int N = nBlk * nThx;
int sizeD = (N + COEF_NBR) * sizeof(int);
int sizeO = N * sizeof(int);
int i, j; //my loop iterators
int result;
 // Alloc space for device copies of D, O
 hipMalloc((void **)&d_D, sizeD);
 hipMalloc((void **)&d_O, sizeO);

 // Alloc space for host copies of D, O and setup input values
 h_D = (int *)malloc(sizeD); random_ints(h_D, N+COEF_NBR);
 h_O = (int *)malloc(sizeO);

 // Copy input data to device
 hipMemcpy(d_D, h_D, sizeD, hipMemcpyHostToDevice);

hipEvent_t start, stop;
float time;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0);

 // Launch add() kernel on GPU with N threads
 fir<<<nBlk,nThx>>>(d_D, d_O);


hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time, start, stop);
printf("GPU time is %f ms\n", time);

 // Copy result back to host
 hipMemcpy(h_O, d_O, sizeO, hipMemcpyDeviceToHost);
 //Checkup
 for(i=0; i<N; i++){
 result=0;
 for(j=0; j<COEF_NBR; j++)
 result += h_C[j] * h_D[i + j];
 if( result != h_O[i] ){
 printf("Error at %d. %d != %d\n", i, h_O[i], result);
 goto cleanup;
 }
 }
 printf("Success\n");
cleanup:
 // Cleanup
 free(h_D); free(h_O);
 hipFree(d_D); hipFree(d_O);
 return 0;
}



