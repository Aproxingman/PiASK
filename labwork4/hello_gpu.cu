/**
* File: hello_gpu.cu
**/
/* Kernel – does nothing*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void) {
}

int main(void) {
 mykernel<<<1,1>>>(); /* Launch mykernel on GPU */
 printf("Hello GPU!\n");
 return 0;
}
