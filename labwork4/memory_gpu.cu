
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c) {
int index = threadIdx.x + blockIdx.x + blockDim.x;
 c[index] = a[index] + b[index];
}


void random_ints(int* a, int N){
int i;
for (i = 0; i < N; i++){
a[i] = (int)(rand() / (RAND_MAX + 1.0) *10.0);
}

}



int main(void) {
int *a, *b, *c; // host copies of a, b, c
int *d_a, *d_b, *d_c; // device copies of a, b, c
int nBlk = 64;
int nThx = 128;
int N = nBlk * nThx;
int size = N * sizeof(int);
int i; //iterator

 // Alloc space for device copies of a, b, c
 hipMalloc((void **)&d_a, size);
 hipMalloc((void **)&d_b, size);
 hipMalloc((void **)&d_c, size);

 // Alloc space for host copies of a, b, c and setup input values
 a = (int *)malloc(size); random_ints(a, N);
 b = (int *)malloc(size); random_ints(b, N);
 c = (int *)malloc(size);

 // Copy inputs to device
 hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
 hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

 // Launch add() kernel on GPU with N threads
 add<<<nBlk,nThx>>>(d_a, d_b, d_c);

 // Copy result back to host
 hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

for ( i=0;i<N;i++) {
printf("a[%d]=%d , b[%d]=%d, c[%d]=%d\n",i,a[i],i,b[i],i,c[i]);
}


 // Cleanup
 free(a); free(b); free(c);
 hipFree(d_a); hipFree(d_b); hipFree(d_c);

 return 0;
}

